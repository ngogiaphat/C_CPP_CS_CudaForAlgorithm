#include "hip/hip_runtime.h"
// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}        
// Host code
int main()
{
    int N = ...;
    size_t size = N * sizeof(float);
    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);
    // Initialize input vectors
    ...
    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);
    //Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =
            (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);   
    // Free host memory..
}
Linear memory can also be allocated through hipMallocPitch() and hipMalloc3D(). These functions are recommended for allocations of 2D or 3D arrays as it makes sure that the allocation is appropriately padded to meet the alignment requirements described in Device Memory Accesses, therefore ensuring best performance when accessing the row addresses or performing copies between 2D arrays and other regions of device memory (using the hipMemcpy2D() and hipMemcpy3D() functions). The returned pitch (or stride) must be used to access array elements. The following code sample allocates a width x height 2D array of floating-point values and shows how to loop over the array elements in device code:
// Host code
int width = 64, height = 64;
float* devPtr;
size_t pitch;
hipMallocPitch(&devPtr, &pitch,
                width * sizeof(float), height);
MyKernel<<<100, 512>>>(devPtr, pitch, width, height);
// Device code
__global__ void MyKernel(float* devPtr,
                         size_t pitch, int width, int height)
{
    for (int r = 0; r < height; ++r) 
    {
        float* row = (float*)((char*)devPtr + r * pitch);
        for (int c = 0; c < width; ++c) 
        {
            float element = row[c];
        }
    }
}