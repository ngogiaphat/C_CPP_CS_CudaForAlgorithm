int driverVersion = 0;
int deviceSupportsMemoryPools = 0;
int deviceSupportsMemoryNodes = 0;
hipDriverGetVersion(&driverVersion);
if (driverVersion >= 11020) 
{
     // avoid invalid value error in hipDeviceGetAttribute
    hipDeviceGetAttribute(&deviceSupportsMemoryPools, hipDeviceAttributeMemoryPoolsSupported, device);
}
deviceSupportsMemoryNodes = (driverVersion >= 11040) && (deviceSupportsMemoryPools != 0);