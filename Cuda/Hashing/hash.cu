#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void hashFunction(const char* input, int* output){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    output[tid] = input[tid] % 256;
}
int main(){
    const int size = 256;
    const int threadsPerBlock = 256;
    const int numBlocks = size / threadsPerBlock;
    char input[size] = "Hello, CUDA!";
    int output[size];
    char* dev_input;
    int* dev_output;
    hipMalloc((void**)&dev_input, size * sizeof(char));
    hipMalloc((void**)&dev_output, size * sizeof(int));
    hipMemcpy(dev_input, input, size * sizeof(char), hipMemcpyHostToDevice);
    hashFunction<<<numBlocks, threadsPerBlock>>>(dev_input, dev_output);
    hipMemcpy(output, dev_output, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_input);
    hipFree(dev_output);
    for(int i = 0; i < size; i++){
        printf("%d ", output[i]);
    }
    printf("\n");
    return 0;
}