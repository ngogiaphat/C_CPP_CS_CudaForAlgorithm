#include<hip/hip_runtime.h>
#include<iostream>
__global__ void addArrays(int* a, int* b, int* c, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size){
        c[tid] = a[tid] + b[tid];
    }
}
int main(){
    int size = 1000;
    int* a, * b, * c;
    int* gpu_a, * gpu_b, * gpu_c;
    // Allocate memory on the host
    a = new int[size];
    b = new int[size];
    c = new int[size];
    // Initialize input arrays
    for(int i = 0; i < size; i++){
        a[i] = i;
        b[i] = size - i;
    }
    // Allocate memory on the GPU
    hipMalloc((void**)&gpu_a, size * sizeof(int));
    hipMalloc((void**)&gpu_b, size * sizeof(int));
    hipMalloc((void**)&gpu_c, size * sizeof(int));
    // Copy input data from host to GPU
    hipMemcpy(gpu_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    // Launch kernel on GPU
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    addArrays<<<numBlocks, blockSize>>>(gpu_a, gpu_b, gpu_c, size);
    // Copy result from GPU to host
    hipMemcpy(c, gpu_c, size * sizeof(int), hipMemcpyDeviceToHost);
    // Print result
    for(int i = 0; i < size; i++){
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;
    // Free memory
    delete[] a;
    delete[] b;
    delete[] c;
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
    return 0;
}