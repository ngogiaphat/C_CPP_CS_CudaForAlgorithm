int driverVersion = 0;
int deviceSupportsMemoryPools = 0;
int poolSupportedHandleTypes = 0;
hipDriverGetVersion(&driverVersion);
if (driverVersion >= 11020) {
    hipDeviceGetAttribute(&deviceSupportsMemoryPools,
                           hipDeviceAttributeMemoryPoolsSupported, device);
}
if (deviceSupportsMemoryPools != 0) {
    // `device` supports the Stream Ordered Memory Allocator
}

if (driverVersion >= 11030) {
    hipDeviceGetAttribute(&poolSupportedHandleTypes,
              cudaDevAttrMemoryPoolSupportedHandleTypes, device);
}
if (poolSupportedHandleTypes & hipMemHandleTypePosixFileDescriptor) {
   // Pools on the specified device can be created with posix file descriptor-based IPC
}